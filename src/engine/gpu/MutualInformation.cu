#include "hip/hip_runtime.h"
/*
 * This file is part of MPI3SNP.
 * Copyright (C) 2014 - 2017 by Jorge González
 * Copyright (C) 2018 by Christian Ponte
 *
 * MPI3SNP is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MPI3SNP is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MPI3SNP. If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * @file gpu/EntropySearch.cu
 * @author Jorge González
 * @author Christian Ponte
 * @date 1 March 2018
 *
 * @brief EntropySearch class members implementation.
 */

#include "MutualInformation.h"
#include <float.h>
#include <algorithm>

/*number of streaming processor scale*/
#define NUM_TH_PER_BLOCK    16

__constant__ float _invInds;
__constant__ float _entY;
__constant__ float _MAX_FLOAT;

static __device__ uint32_t
_devpopcount(uint32_t
v) {
//uint32_t u;
//u = v - ((v>>1) & 033333333333) - ((v>>2) & 011111111111);
// return ((u + (u>>3)) & 030707070707) % 63;
return
__popc(v);
}

static __global__ void _kernelDoubleTable(uint64_t numPairs, uint32_t numSNPs, uint16_t numEntriesCases,
                                          uint16_t numEntriesCtrls, uint2 *devIds, uint32_t *dev0Cases,
                                          uint32_t *dev1Cases, uint32_t *dev2Cases, uint32_t *dev0Ctrls,
                                          uint32_t *dev1Ctrls, uint32_t *dev2Ctrls,
                                          GPUDoubleContTable *doubleTables) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x; /*global id*/

    if (gid >= numPairs) {
        return;
    }

    uint32_t
    myId1 = devIds[gid].x;
    uint32_t
    myId2 = devIds[gid].y;
    uint32_t
    entry1 = myId1;
    uint32_t
    entry2 = myId2;

    GPUDoubleContTable *table = &doubleTables[gid];

    for (int i = 0; i < numEntriesCases; i++, entry1 += numSNPs, entry2 += numSNPs) {
        table->_cases00[i] = dev0Cases[entry1] & dev0Cases[entry2];
        table->_cases01[i] = dev0Cases[entry1] & dev1Cases[entry2];
        table->_cases02[i] = dev0Cases[entry1] & dev2Cases[entry2];
        table->_cases10[i] = dev1Cases[entry1] & dev0Cases[entry2];
        table->_cases11[i] = dev1Cases[entry1] & dev1Cases[entry2];
        table->_cases12[i] = dev1Cases[entry1] & dev2Cases[entry2];
        table->_cases20[i] = dev2Cases[entry1] & dev0Cases[entry2];
        table->_cases21[i] = dev2Cases[entry1] & dev1Cases[entry2];
        table->_cases22[i] = dev2Cases[entry1] & dev2Cases[entry2];
    }

    entry1 = myId1;
    entry2 = myId2;

    for (int i = 0; i < numEntriesCtrls; i++, entry1 += numSNPs, entry2 += numSNPs) {
        table->_ctrls00[i] = dev0Ctrls[entry1] & dev0Ctrls[entry2];
        table->_ctrls01[i] = dev0Ctrls[entry1] & dev1Ctrls[entry2];
        table->_ctrls02[i] = dev0Ctrls[entry1] & dev2Ctrls[entry2];
        table->_ctrls10[i] = dev1Ctrls[entry1] & dev0Ctrls[entry2];
        table->_ctrls11[i] = dev1Ctrls[entry1] & dev1Ctrls[entry2];
        table->_ctrls12[i] = dev1Ctrls[entry1] & dev2Ctrls[entry2];
        table->_ctrls20[i] = dev2Ctrls[entry1] & dev0Ctrls[entry2];
        table->_ctrls21[i] = dev2Ctrls[entry1] & dev1Ctrls[entry2];
        table->_ctrls22[i] = dev2Ctrls[entry1] & dev2Ctrls[entry2];
    }
}

static __global__ void _kernelTripleMI(uint64_t numPairs, uint32_t numSNPs,
                                       uint16_t numEntriesCases, uint16_t numEntriesCtrls, uint2 *devIds,
                                       uint32_t *dev0Cases, uint32_t *dev1Cases, uint32_t *dev2Cases,
                                       uint32_t *dev0Ctrls, uint32_t *dev1Ctrls, uint32_t *dev2Ctrls,
                                       GPUDoubleContTable *devDoubleTables, uint16_t numOutputs,
                                       float *devMIValues, uint3 *devMiIds) {

    extern __shared__ uint32_t
    sharedMem[];
    uint32_t * cases00 = sharedMem;
    uint32_t * cases01 = &sharedMem[numEntriesCases];
    uint32_t * cases02 = &sharedMem[2 * numEntriesCases];
    uint32_t * cases10 = &sharedMem[3 * numEntriesCases];
    uint32_t * cases11 = &sharedMem[4 * numEntriesCases];
    uint32_t * cases12 = &sharedMem[5 * numEntriesCases];
    uint32_t * cases20 = &sharedMem[6 * numEntriesCases];
    uint32_t * cases21 = &sharedMem[7 * numEntriesCases];
    uint32_t * cases22 = &sharedMem[8 * numEntriesCases];
    uint32_t * ctrls00 = &sharedMem[9 * numEntriesCases];
    uint32_t * ctrls01 = &sharedMem[9 * numEntriesCases + numEntriesCtrls];
    uint32_t * ctrls02 = &sharedMem[9 * numEntriesCases + 2 * numEntriesCtrls];
    uint32_t * ctrls10 = &sharedMem[9 * numEntriesCases + 3 * numEntriesCtrls];
    uint32_t * ctrls11 = &sharedMem[9 * numEntriesCases + 4 * numEntriesCtrls];
    uint32_t * ctrls12 = &sharedMem[9 * numEntriesCases + 5 * numEntriesCtrls];
    uint32_t * ctrls20 = &sharedMem[9 * numEntriesCases + 6 * numEntriesCtrls];
    uint32_t * ctrls21 = &sharedMem[9 * numEntriesCases + 7 * numEntriesCtrls];
    uint32_t * ctrls22 = &sharedMem[9 * numEntriesCases + 8 * numEntriesCtrls];
    uint32_t * shMIId = &sharedMem[9 * (numEntriesCases + numEntriesCtrls)];
    float *shMIValues = (float *) &sharedMem[9 * (numEntriesCases + numEntriesCtrls) + blockDim.x * numOutputs];

    uint32_t * myOutIds = &shMIId[threadIdx.x * numOutputs];
    float *myOutValues = &shMIValues[threadIdx.x * numOutputs];

    uint16_t numEntriesWithMI = 0;
    float minMI = _MAX_FLOAT;
    uint16_t minMIPos = 0;

    // Copy the information of the double contingency table to shared memory
    GPUDoubleContTable *table = &devDoubleTables[blockIdx.x];

    for (int iter = threadIdx.x; iter < numEntriesCases; iter += blockDim.x) {
        cases00[iter] = table->_cases00[iter];
        cases01[iter] = table->_cases01[iter];
        cases02[iter] = table->_cases02[iter];
        cases10[iter] = table->_cases10[iter];
        cases11[iter] = table->_cases11[iter];
        cases12[iter] = table->_cases12[iter];
        cases20[iter] = table->_cases20[iter];
        cases21[iter] = table->_cases21[iter];
        cases22[iter] = table->_cases22[iter];
    }
    for (int iter = threadIdx.x; iter < numEntriesCtrls; iter += blockDim.x) {
        ctrls00[iter] = table->_ctrls00[iter];
        ctrls01[iter] = table->_ctrls01[iter];
        ctrls02[iter] = table->_ctrls02[iter];
        ctrls10[iter] = table->_ctrls10[iter];
        ctrls11[iter] = table->_ctrls11[iter];
        ctrls12[iter] = table->_ctrls12[iter];
        ctrls20[iter] = table->_ctrls20[iter];
        ctrls21[iter] = table->_ctrls21[iter];
        ctrls22[iter] = table->_ctrls22[iter];
    }

    __syncthreads();

    uint16_t tripleCases[27];
    uint16_t tripleCtrls[27];

    uint32_t
    myId1 = devIds[blockIdx.x].x;
    uint32_t
    myId2 = devIds[blockIdx.x].y;
    uint32_t
    iterId3 = myId2 + threadIdx.x + 1;

    uint32_t
    aux;
    uint32_t
    auxSNP3Value;

    // Each thread computes several triples using the same pair
    for (; iterId3 < numSNPs; iterId3 += blockDim.x) {
        // Starts creating the contingency table
        for (int i = 0; i < 27; i++) {
            tripleCases[i] = 0;
            tripleCtrls[i] = 0;
        }

        for (int i = 0; i < numEntriesCases; i++) {
            auxSNP3Value = dev0Cases[iterId3 + i * numSNPs];

            aux = cases00[i] & auxSNP3Value;
            tripleCases[0] += _devpopcount(aux);

            aux = cases01[i] & auxSNP3Value;
            tripleCases[1] += _devpopcount(aux);

            aux = cases02[i] & auxSNP3Value;
            tripleCases[2] += _devpopcount(aux);

            aux = cases10[i] & auxSNP3Value;
            tripleCases[3] += _devpopcount(aux);

            aux = cases11[i] & auxSNP3Value;
            tripleCases[4] += _devpopcount(aux);

            aux = cases12[i] & auxSNP3Value;
            tripleCases[5] += _devpopcount(aux);

            aux = cases20[i] & auxSNP3Value;
            tripleCases[6] += _devpopcount(aux);

            aux = cases21[i] & auxSNP3Value;
            tripleCases[7] += _devpopcount(aux);

            aux = cases22[i] & auxSNP3Value;
            tripleCases[8] += _devpopcount(aux);

            auxSNP3Value = dev1Cases[iterId3 + i * numSNPs];

            aux = cases00[i] & auxSNP3Value;
            tripleCases[9] += _devpopcount(aux);

            aux = cases01[i] & auxSNP3Value;
            tripleCases[10] += _devpopcount(aux);

            aux = cases02[i] & auxSNP3Value;
            tripleCases[11] += _devpopcount(aux);

            aux = cases10[i] & auxSNP3Value;
            tripleCases[12] += _devpopcount(aux);

            aux = cases11[i] & auxSNP3Value;
            tripleCases[13] += _devpopcount(aux);

            aux = cases12[i] & auxSNP3Value;
            tripleCases[14] += _devpopcount(aux);

            aux = cases20[i] & auxSNP3Value;
            tripleCases[15] += _devpopcount(aux);

            aux = cases21[i] & auxSNP3Value;
            tripleCases[16] += _devpopcount(aux);

            aux = cases22[i] & auxSNP3Value;
            tripleCases[17] += _devpopcount(aux);

            auxSNP3Value = dev2Cases[iterId3 + i * numSNPs];

            aux = cases00[i] & auxSNP3Value;
            tripleCases[18] += _devpopcount(aux);

            aux = cases01[i] & auxSNP3Value;
            tripleCases[19] += _devpopcount(aux);

            aux = cases02[i] & auxSNP3Value;
            tripleCases[20] += _devpopcount(aux);

            aux = cases10[i] & auxSNP3Value;
            tripleCases[21] += _devpopcount(aux);

            aux = cases11[i] & auxSNP3Value;
            tripleCases[22] += _devpopcount(aux);

            aux = cases12[i] & auxSNP3Value;
            tripleCases[23] += _devpopcount(aux);

            aux = cases20[i] & auxSNP3Value;
            tripleCases[24] += _devpopcount(aux);

            aux = cases21[i] & auxSNP3Value;
            tripleCases[25] += _devpopcount(aux);

            aux = cases22[i] & auxSNP3Value;
            tripleCases[26] += _devpopcount(aux);
        }

        for (int i = 0; i < numEntriesCtrls; i++) {
            auxSNP3Value = dev0Ctrls[iterId3 + i * numSNPs];

            aux = ctrls00[i] & auxSNP3Value;
            tripleCtrls[0] += _devpopcount(aux);

            aux = ctrls01[i] & auxSNP3Value;
            tripleCtrls[1] += _devpopcount(aux);

            aux = ctrls02[i] & auxSNP3Value;
            tripleCtrls[2] += _devpopcount(aux);

            aux = ctrls10[i] & auxSNP3Value;
            tripleCtrls[3] += _devpopcount(aux);

            aux = ctrls11[i] & auxSNP3Value;
            tripleCtrls[4] += _devpopcount(aux);

            aux = ctrls12[i] & auxSNP3Value;
            tripleCtrls[5] += _devpopcount(aux);

            aux = ctrls20[i] & auxSNP3Value;
            tripleCtrls[6] += _devpopcount(aux);

            aux = ctrls21[i] & auxSNP3Value;
            tripleCtrls[7] += _devpopcount(aux);

            aux = ctrls22[i] & auxSNP3Value;
            tripleCtrls[8] += _devpopcount(aux);

            auxSNP3Value = dev1Ctrls[iterId3 + i * numSNPs];

            aux = ctrls00[i] & auxSNP3Value;
            tripleCtrls[9] += _devpopcount(aux);

            aux = ctrls01[i] & auxSNP3Value;
            tripleCtrls[10] += _devpopcount(aux);

            aux = ctrls02[i] & auxSNP3Value;
            tripleCtrls[11] += _devpopcount(aux);

            aux = ctrls10[i] & auxSNP3Value;
            tripleCtrls[12] += _devpopcount(aux);

            aux = ctrls11[i] & auxSNP3Value;
            tripleCtrls[13] += _devpopcount(aux);

            aux = ctrls12[i] & auxSNP3Value;
            tripleCtrls[14] += _devpopcount(aux);

            aux = ctrls20[i] & auxSNP3Value;
            tripleCtrls[15] += _devpopcount(aux);

            aux = ctrls21[i] & auxSNP3Value;
            tripleCtrls[16] += _devpopcount(aux);

            aux = ctrls22[i] & auxSNP3Value;
            tripleCtrls[17] += _devpopcount(aux);

            auxSNP3Value = dev2Ctrls[iterId3 + i * numSNPs];

            aux = ctrls00[i] & auxSNP3Value;
            tripleCtrls[18] += _devpopcount(aux);

            aux = ctrls01[i] & auxSNP3Value;
            tripleCtrls[19] += _devpopcount(aux);

            aux = ctrls02[i] & auxSNP3Value;
            tripleCtrls[20] += _devpopcount(aux);

            aux = ctrls10[i] & auxSNP3Value;
            tripleCtrls[21] += _devpopcount(aux);

            aux = ctrls11[i] & auxSNP3Value;
            tripleCtrls[22] += _devpopcount(aux);

            aux = ctrls12[i] & auxSNP3Value;
            tripleCtrls[23] += _devpopcount(aux);

            aux = ctrls20[i] & auxSNP3Value;
            tripleCtrls[24] += _devpopcount(aux);

            aux = ctrls21[i] & auxSNP3Value;
            tripleCtrls[25] += _devpopcount(aux);

            aux = ctrls22[i] & auxSNP3Value;
            tripleCtrls[26] += _devpopcount(aux);
        }

        // Calculate the MI with the values of the contingency table
        float entX = 0.0;
        float entAll = 0.0;
        float pCase, pCtrl;

        for (int i = 0; i < 27; i++) {
            pCase = tripleCases[i] * _invInds;
            if (pCase != 0.0) {
                entAll -= pCase * log2(pCase);
            }

            pCtrl = tripleCtrls[i] * _invInds;
            if (pCtrl != 0.0) {
                entAll -= pCtrl * log2(pCtrl);
            }

            pCase += pCtrl;
            if (pCase != 0.0) {
                entX -= pCase * log2(pCase);
            }
        }

        // The result of the MI is now in entX
        entX += _entY - entAll;

#ifdef DEBUG
        printf("Thread %d in block %d: MI for triple (%u, %u, %u) is %f\n", threadIdx.x,
                blockIdx.x, myId1, myId2, iterId3, entX);
#endif

        // Now include the value in the output list if it is high enough
        // There are empty values in the array
        if (numEntriesWithMI < numOutputs) {
            myOutIds[numEntriesWithMI] = iterId3;
            myOutValues[numEntriesWithMI] = entX;

            // If this is the minimum value of the array
            if (entX < minMI) {
                minMI = entX;
                minMIPos = numEntriesWithMI;
            }

            numEntriesWithMI++;
        } else if (entX > minMI) { // The value must be inserted
            myOutIds[minMIPos] = iterId3;
            myOutValues[minMIPos] = entX;

            // Find the new minimum
            minMIPos = 0;
            minMI = myOutIds[0];
            for (int i = 1; i < numOutputs; i++) {
                if (myOutValues[i] < minMI) {
                    minMI = myOutValues[i];
                    minMIPos = i;
                }
            }
        }
    }

    // The thread has a list of numOutputs with the highest values
    // Complete the list just in case there are no so many values in total
    for (int i = numEntriesWithMI; i < numOutputs; i++) {
        myOutIds[i] = 0;
        myOutValues[i] = 0.0;
        minMI = 0.0;
        minMIPos = i;
    }

#ifdef DEBUG
    printf("Before reducing thread %d of block %d: %u (%f), %u (%f), %u (%f), %u (%f)\n",
                threadIdx.x, blockIdx.x,
                myOutIds[0], myOutValues[0], myOutIds[1], myOutValues[1],
                myOutIds[2], myOutValues[2], myOutIds[3], myOutValues[3]);
#endif

    float *remoteOutValues;
    uint32_t * remoteOutIds;

    // Perform the reduction of the lists of the block of threads
    // Each reduction obtains the numOutputs highest elements of two threads
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        __syncthreads();
        if (threadIdx.x < stride) {
            // Each thread has its own minimum so we only need to compare the numOutputs values of the other thread
            remoteOutValues = &shMIValues[(threadIdx.x + stride) * numOutputs];
            remoteOutIds = &shMIId[(threadIdx.x + stride) * numOutputs];

            for (int i = 0; i < numOutputs; i++) {
                if (remoteOutValues[i] > minMI) { // The value must be inserted
                    myOutIds[minMIPos] = remoteOutIds[i];
                    myOutValues[minMIPos] = remoteOutValues[i];

                    // Find the new minimum
                    minMIPos = 0;
                    minMI = myOutIds[0];
                    for (int j = 1; j < numOutputs; j++) {
                        if (myOutValues[j] < minMI) {
                            minMI = myOutValues[j];
                            minMIPos = j;
                        }
                    }
                }
            }
#ifdef DEBUG
            printf("With stride %d thread %d of block %d: %u (%f), %u (%f), %u (%f), %u (%f)\n", stride,
                threadIdx.x, blockIdx.x,
                myOutIds[0], myOutValues[0], myOutIds[1], myOutValues[1],
                myOutIds[2], myOutValues[2], myOutIds[3], myOutValues[3]);
#endif
        }
    }

    // Save the output list for the block
    if (!threadIdx.x) {
        uint3 *blockOutIds = &devMiIds[blockIdx.x * numOutputs];
        float *blockOutValues = &devMIValues[blockIdx.x * numOutputs];

        for (int i = 0; i < numOutputs; i++) {
            blockOutIds[i].x = myId1;
            blockOutIds[i].y = myId2;
            blockOutIds[i].z = myOutIds[i];
            blockOutValues[i] = myOutValues[i];
        }
    }
}

static __global__ void _kernelTripleIG(uint64_t numPairs, uint32_t numSNPs,
                                       uint16_t numEntriesCases, uint16_t numEntriesCtrls, uint2 *devIds,
                                       uint32_t *dev0Cases, uint32_t *dev1Cases, uint32_t *dev2Cases,
                                       uint32_t *dev0Ctrls, uint32_t *dev1Ctrls, uint32_t *dev2Ctrls,
                                       GPUDoubleContTable *devDoubleTables, uint16_t numOutputs,
                                       float *devMIValues, uint3 *devMiIds) {

    extern __shared__ uint32_t
    sharedMem[];
    uint32_t * cases00 = sharedMem;
    uint32_t * cases01 = &sharedMem[numEntriesCases];
    uint32_t * cases02 = &sharedMem[2 * numEntriesCases];
    uint32_t * cases10 = &sharedMem[3 * numEntriesCases];
    uint32_t * cases11 = &sharedMem[4 * numEntriesCases];
    uint32_t * cases12 = &sharedMem[5 * numEntriesCases];
    uint32_t * cases20 = &sharedMem[6 * numEntriesCases];
    uint32_t * cases21 = &sharedMem[7 * numEntriesCases];
    uint32_t * cases22 = &sharedMem[8 * numEntriesCases];
    uint32_t * ctrls00 = &sharedMem[9 * numEntriesCases];
    uint32_t * ctrls01 = &sharedMem[9 * numEntriesCases + numEntriesCtrls];
    uint32_t * ctrls02 = &sharedMem[9 * numEntriesCases + 2 * numEntriesCtrls];
    uint32_t * ctrls10 = &sharedMem[9 * numEntriesCases + 3 * numEntriesCtrls];
    uint32_t * ctrls11 = &sharedMem[9 * numEntriesCases + 4 * numEntriesCtrls];
    uint32_t * ctrls12 = &sharedMem[9 * numEntriesCases + 5 * numEntriesCtrls];
    uint32_t * ctrls20 = &sharedMem[9 * numEntriesCases + 6 * numEntriesCtrls];
    uint32_t * ctrls21 = &sharedMem[9 * numEntriesCases + 7 * numEntriesCtrls];
    uint32_t * ctrls22 = &sharedMem[9 * numEntriesCases + 8 * numEntriesCtrls];
    uint32_t * shMIId = &sharedMem[9 * (numEntriesCases + numEntriesCtrls)];
    float *shMIValues = (float *) &sharedMem[9 * (numEntriesCases + numEntriesCtrls) + blockDim.x * numOutputs];

    uint32_t * myOutIds = &shMIId[threadIdx.x * numOutputs];
    float *myOutValues = &shMIValues[threadIdx.x * numOutputs];

    uint16_t numEntriesWithMI = 0;
    float minMI = _MAX_FLOAT;
    uint16_t minMIPos = 0;

    // Copy the information of the double contingency table to shared memory
    GPUDoubleContTable *table = &devDoubleTables[blockIdx.x];

    for (int iter = threadIdx.x; iter < numEntriesCases; iter += blockDim.x) {
        cases00[iter] = table->_cases00[iter];
        cases01[iter] = table->_cases01[iter];
        cases02[iter] = table->_cases02[iter];
        cases10[iter] = table->_cases10[iter];
        cases11[iter] = table->_cases11[iter];
        cases12[iter] = table->_cases12[iter];
        cases20[iter] = table->_cases20[iter];
        cases21[iter] = table->_cases21[iter];
        cases22[iter] = table->_cases22[iter];
    }
    for (int iter = threadIdx.x; iter < numEntriesCtrls; iter += blockDim.x) {
        ctrls00[iter] = table->_ctrls00[iter];
        ctrls01[iter] = table->_ctrls01[iter];
        ctrls02[iter] = table->_ctrls02[iter];
        ctrls10[iter] = table->_ctrls10[iter];
        ctrls11[iter] = table->_ctrls11[iter];
        ctrls12[iter] = table->_ctrls12[iter];
        ctrls20[iter] = table->_ctrls20[iter];
        ctrls21[iter] = table->_ctrls21[iter];
        ctrls22[iter] = table->_ctrls22[iter];
    }

    __syncthreads();

    uint16_t tripleCases[27];
    uint16_t tripleCtrls[27];

    uint32_t
    myId1 = devIds[blockIdx.x].x;
    uint32_t
    myId2 = devIds[blockIdx.x].y;
    uint32_t
    iterId3 = myId2 + threadIdx.x + 1;

    uint32_t
    aux;
    uint32_t
    auxSNP3Value;

    // Each thread computes several triples using the same pair
    for (; iterId3 < numSNPs; iterId3 += blockDim.x) {
        // Starts creating the contingency table
        for (int i = 0; i < 27; i++) {
            tripleCases[i] = 0;
            tripleCtrls[i] = 0;
        }

        for (int i = 0; i < numEntriesCases; i++) {
            auxSNP3Value = dev0Cases[iterId3 + i * numSNPs];

            aux = cases00[i] & auxSNP3Value;
            tripleCases[0] += _devpopcount(aux);

            aux = cases01[i] & auxSNP3Value;
            tripleCases[1] += _devpopcount(aux);

            aux = cases02[i] & auxSNP3Value;
            tripleCases[2] += _devpopcount(aux);

            aux = cases10[i] & auxSNP3Value;
            tripleCases[3] += _devpopcount(aux);

            aux = cases11[i] & auxSNP3Value;
            tripleCases[4] += _devpopcount(aux);

            aux = cases12[i] & auxSNP3Value;
            tripleCases[5] += _devpopcount(aux);

            aux = cases20[i] & auxSNP3Value;
            tripleCases[6] += _devpopcount(aux);

            aux = cases21[i] & auxSNP3Value;
            tripleCases[7] += _devpopcount(aux);

            aux = cases22[i] & auxSNP3Value;
            tripleCases[8] += _devpopcount(aux);

            auxSNP3Value = dev1Cases[iterId3 + i * numSNPs];

            aux = cases00[i] & auxSNP3Value;
            tripleCases[9] += _devpopcount(aux);

            aux = cases01[i] & auxSNP3Value;
            tripleCases[10] += _devpopcount(aux);

            aux = cases02[i] & auxSNP3Value;
            tripleCases[11] += _devpopcount(aux);

            aux = cases10[i] & auxSNP3Value;
            tripleCases[12] += _devpopcount(aux);

            aux = cases11[i] & auxSNP3Value;
            tripleCases[13] += _devpopcount(aux);

            aux = cases12[i] & auxSNP3Value;
            tripleCases[14] += _devpopcount(aux);

            aux = cases20[i] & auxSNP3Value;
            tripleCases[15] += _devpopcount(aux);

            aux = cases21[i] & auxSNP3Value;
            tripleCases[16] += _devpopcount(aux);

            aux = cases22[i] & auxSNP3Value;
            tripleCases[17] += _devpopcount(aux);

            auxSNP3Value = dev2Cases[iterId3 + i * numSNPs];

            aux = cases00[i] & auxSNP3Value;
            tripleCases[18] += _devpopcount(aux);

            aux = cases01[i] & auxSNP3Value;
            tripleCases[19] += _devpopcount(aux);

            aux = cases02[i] & auxSNP3Value;
            tripleCases[20] += _devpopcount(aux);

            aux = cases10[i] & auxSNP3Value;
            tripleCases[21] += _devpopcount(aux);

            aux = cases11[i] & auxSNP3Value;
            tripleCases[22] += _devpopcount(aux);

            aux = cases12[i] & auxSNP3Value;
            tripleCases[23] += _devpopcount(aux);

            aux = cases20[i] & auxSNP3Value;
            tripleCases[24] += _devpopcount(aux);

            aux = cases21[i] & auxSNP3Value;
            tripleCases[25] += _devpopcount(aux);

            aux = cases22[i] & auxSNP3Value;
            tripleCases[26] += _devpopcount(aux);
        }

        for (int i = 0; i < numEntriesCtrls; i++) {
            auxSNP3Value = dev0Ctrls[iterId3 + i * numSNPs];

            aux = ctrls00[i] & auxSNP3Value;
            tripleCtrls[0] += _devpopcount(aux);

            aux = ctrls01[i] & auxSNP3Value;
            tripleCtrls[1] += _devpopcount(aux);

            aux = ctrls02[i] & auxSNP3Value;
            tripleCtrls[2] += _devpopcount(aux);

            aux = ctrls10[i] & auxSNP3Value;
            tripleCtrls[3] += _devpopcount(aux);

            aux = ctrls11[i] & auxSNP3Value;
            tripleCtrls[4] += _devpopcount(aux);

            aux = ctrls12[i] & auxSNP3Value;
            tripleCtrls[5] += _devpopcount(aux);

            aux = ctrls20[i] & auxSNP3Value;
            tripleCtrls[6] += _devpopcount(aux);

            aux = ctrls21[i] & auxSNP3Value;
            tripleCtrls[7] += _devpopcount(aux);

            aux = ctrls22[i] & auxSNP3Value;
            tripleCtrls[8] += _devpopcount(aux);

            auxSNP3Value = dev1Ctrls[iterId3 + i * numSNPs];

            aux = ctrls00[i] & auxSNP3Value;
            tripleCtrls[9] += _devpopcount(aux);

            aux = ctrls01[i] & auxSNP3Value;
            tripleCtrls[10] += _devpopcount(aux);

            aux = ctrls02[i] & auxSNP3Value;
            tripleCtrls[11] += _devpopcount(aux);

            aux = ctrls10[i] & auxSNP3Value;
            tripleCtrls[12] += _devpopcount(aux);

            aux = ctrls11[i] & auxSNP3Value;
            tripleCtrls[13] += _devpopcount(aux);

            aux = ctrls12[i] & auxSNP3Value;
            tripleCtrls[14] += _devpopcount(aux);

            aux = ctrls20[i] & auxSNP3Value;
            tripleCtrls[15] += _devpopcount(aux);

            aux = ctrls21[i] & auxSNP3Value;
            tripleCtrls[16] += _devpopcount(aux);

            aux = ctrls22[i] & auxSNP3Value;
            tripleCtrls[17] += _devpopcount(aux);

            auxSNP3Value = dev2Ctrls[iterId3 + i * numSNPs];

            aux = ctrls00[i] & auxSNP3Value;
            tripleCtrls[18] += _devpopcount(aux);

            aux = ctrls01[i] & auxSNP3Value;
            tripleCtrls[19] += _devpopcount(aux);

            aux = ctrls02[i] & auxSNP3Value;
            tripleCtrls[20] += _devpopcount(aux);

            aux = ctrls10[i] & auxSNP3Value;
            tripleCtrls[21] += _devpopcount(aux);

            aux = ctrls11[i] & auxSNP3Value;
            tripleCtrls[22] += _devpopcount(aux);

            aux = ctrls12[i] & auxSNP3Value;
            tripleCtrls[23] += _devpopcount(aux);

            aux = ctrls20[i] & auxSNP3Value;
            tripleCtrls[24] += _devpopcount(aux);

            aux = ctrls21[i] & auxSNP3Value;
            tripleCtrls[25] += _devpopcount(aux);

            aux = ctrls22[i] & auxSNP3Value;
            tripleCtrls[26] += _devpopcount(aux);
        }

        float casex0 = 0.0;
        float casex1 = 0.0;
        float casex2 = 0.0;
        float casey0 = 0.0;
        float casey1 = 0.0;
        float casey2 = 0.0;
        float casez0 = 0.0;
        float casez1 = 0.0;
        float casez2 = 0.0;
        float casexy00 = 0.0;
        float casexy01 = 0.0;
        float casexy02 = 0.0;
        float casexy10 = 0.0;
        float casexy11 = 0.0;
        float casexy12 = 0.0;
        float casexy20 = 0.0;
        float casexy21 = 0.0;
        float casexy22 = 0.0;
        float casexz00 = 0.0;
        float casexz01 = 0.0;
        float casexz02 = 0.0;
        float casexz10 = 0.0;
        float casexz11 = 0.0;
        float casexz12 = 0.0;
        float casexz20 = 0.0;
        float casexz21 = 0.0;
        float casexz22 = 0.0;
        float caseyz00 = 0.0;
        float caseyz01 = 0.0;
        float caseyz02 = 0.0;
        float caseyz10 = 0.0;
        float caseyz11 = 0.0;
        float caseyz12 = 0.0;
        float caseyz20 = 0.0;
        float caseyz21 = 0.0;
        float caseyz22 = 0.0;
        float ctrlx0 = 0.0;
        float ctrlx1 = 0.0;
        float ctrlx2 = 0.0;
        float ctrly0 = 0.0;
        float ctrly1 = 0.0;
        float ctrly2 = 0.0;
        float ctrlz0 = 0.0;
        float ctrlz1 = 0.0;
        float ctrlz2 = 0.0;
        float ctrlxy00 = 0.0;
        float ctrlxy01 = 0.0;
        float ctrlxy02 = 0.0;
        float ctrlxy10 = 0.0;
        float ctrlxy11 = 0.0;
        float ctrlxy12 = 0.0;
        float ctrlxy20 = 0.0;
        float ctrlxy21 = 0.0;
        float ctrlxy22 = 0.0;
        float ctrlxz00 = 0.0;
        float ctrlxz01 = 0.0;
        float ctrlxz02 = 0.0;
        float ctrlxz10 = 0.0;
        float ctrlxz11 = 0.0;
        float ctrlxz12 = 0.0;
        float ctrlxz20 = 0.0;
        float ctrlxz21 = 0.0;
        float ctrlxz22 = 0.0;
        float ctrlyz00 = 0.0;
        float ctrlyz01 = 0.0;
        float ctrlyz02 = 0.0;
        float ctrlyz10 = 0.0;
        float ctrlyz11 = 0.0;
        float ctrlyz12 = 0.0;
        float ctrlyz20 = 0.0;
        float ctrlyz21 = 0.0;
        float ctrlyz22 = 0.0;
        float entAll = 0.0, entX = 0.0;
        float pCase, pCtrl, miXYZ;

        // Do it one by one in order to not repeat calculations or do extra if-else
        // Calculate the entropies
        // 000
        pCase = tripleCases[0] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey0 += pCase;
        casez0 += pCase;
        casexy00 += pCase;
        casexz00 += pCase;
        caseyz00 += pCase;

        pCtrl = tripleCtrls[0] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy00 += pCtrl;
        ctrlxz00 += pCtrl;
        ctrlyz00 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 001
        pCase = tripleCases[1] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey0 += pCase;
        casez1 += pCase;
        casexy00 += pCase;
        casexz01 += pCase;
        caseyz01 += pCase;

        pCtrl = tripleCtrls[1] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy00 += pCtrl;
        ctrlxz01 += pCtrl;
        ctrlyz01 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 002
        pCase = tripleCases[2] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey0 += pCase;
        casez2 += pCase;
        casexy00 += pCase;
        casexz02 += pCase;
        caseyz02 += pCase;

        pCtrl = tripleCtrls[2] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy00 += pCtrl;
        ctrlxz02 += pCtrl;
        ctrlyz02 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 010
        pCase = tripleCases[3] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey1 += pCase;
        casez0 += pCase;
        casexy01 += pCase;
        casexz00 += pCase;
        caseyz10 += pCase;

        pCtrl = tripleCtrls[3] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy01 += pCtrl;
        ctrlxz00 += pCtrl;
        ctrlyz10 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 011
        pCase = tripleCases[4] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey1 += pCase;
        casez1 += pCase;
        casexy01 += pCase;
        casexz01 += pCase;
        caseyz11 += pCase;

        pCtrl = tripleCtrls[4] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy01 += pCtrl;
        ctrlxz01 += pCtrl;
        ctrlyz11 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 012
        pCase = tripleCases[5] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey1 += pCase;
        casez2 += pCase;
        casexy01 += pCase;
        casexz02 += pCase;
        caseyz12 += pCase;

        pCtrl = tripleCtrls[5] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy01 += pCtrl;
        ctrlxz02 += pCtrl;
        ctrlyz12 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 020
        pCase = tripleCases[6] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey2 += pCase;
        casez0 += pCase;
        casexy02 += pCase;
        casexz00 += pCase;
        caseyz20 += pCase;

        pCtrl = tripleCtrls[6] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy02 += pCtrl;
        ctrlxz00 += pCtrl;
        ctrlyz20 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 021
        pCase = tripleCases[7] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey2 += pCase;
        casez1 += pCase;
        casexy02 += pCase;
        casexz01 += pCase;
        caseyz21 += pCase;

        pCtrl = tripleCtrls[7] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy02 += pCtrl;
        ctrlxz01 += pCtrl;
        ctrlyz21 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 022
        pCase = tripleCases[8] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex0 += pCase;
        casey2 += pCase;
        casez2 += pCase;
        casexy02 += pCase;
        casexz02 += pCase;
        caseyz22 += pCase;

        pCtrl = tripleCtrls[8] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx0 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy02 += pCtrl;
        ctrlxz02 += pCtrl;
        ctrlyz22 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 100
        pCase = tripleCases[9] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey0 += pCase;
        casez0 += pCase;
        casexy10 += pCase;
        casexz10 += pCase;
        caseyz00 += pCase;

        pCtrl = tripleCtrls[9] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy10 += pCtrl;
        ctrlxz10 += pCtrl;
        ctrlyz00 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 101
        pCase = tripleCases[10] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey0 += pCase;
        casez1 += pCase;
        casexy10 += pCase;
        casexz11 += pCase;
        caseyz01 += pCase;

        pCtrl = tripleCtrls[10] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy10 += pCtrl;
        ctrlxz11 += pCtrl;
        ctrlyz01 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 102
        pCase = tripleCases[11] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey0 += pCase;
        casez2 += pCase;
        casexy10 += pCase;
        casexz12 += pCase;
        caseyz02 += pCase;

        pCtrl = tripleCtrls[11] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy10 += pCtrl;
        ctrlxz12 += pCtrl;
        ctrlyz02 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 110
        pCase = tripleCases[12] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey1 += pCase;
        casez0 += pCase;
        casexy11 += pCase;
        casexz10 += pCase;
        caseyz10 += pCase;

        pCtrl = tripleCtrls[12] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy11 += pCtrl;
        ctrlxz10 += pCtrl;
        ctrlyz10 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 111
        pCase = tripleCases[13] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey1 += pCase;
        casez1 += pCase;
        casexy11 += pCase;
        casexz11 += pCase;
        caseyz11 += pCase;

        pCtrl = tripleCtrls[13] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy11 += pCtrl;
        ctrlxz11 += pCtrl;
        ctrlyz11 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 112
        pCase = tripleCases[14] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey1 += pCase;
        casez2 += pCase;
        casexy11 += pCase;
        casexz12 += pCase;
        caseyz12 += pCase;

        pCtrl = tripleCtrls[14] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy11 += pCtrl;
        ctrlxz12 += pCtrl;
        ctrlyz12 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 120
        pCase = tripleCases[15] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey2 += pCase;
        casez0 += pCase;
        casexy12 += pCase;
        casexz10 += pCase;
        caseyz20 += pCase;

        pCtrl = tripleCtrls[15] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy12 += pCtrl;
        ctrlxz10 += pCtrl;
        ctrlyz20 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 121
        pCase = tripleCases[16] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey2 += pCase;
        casez1 += pCase;
        casexy12 += pCase;
        casexz11 += pCase;
        caseyz21 += pCase;

        pCtrl = tripleCtrls[16] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy12 += pCtrl;
        ctrlxz11 += pCtrl;
        ctrlyz21 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 122
        pCase = tripleCases[17] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex1 += pCase;
        casey2 += pCase;
        casez2 += pCase;
        casexy12 += pCase;
        casexz12 += pCase;
        caseyz22 += pCase;

        pCtrl = tripleCtrls[17] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx1 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy12 += pCtrl;
        ctrlxz12 += pCtrl;
        ctrlyz22 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 200
        pCase = tripleCases[18] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey0 += pCase;
        casez0 += pCase;
        casexy20 += pCase;
        casexz20 += pCase;
        caseyz00 += pCase;

        pCtrl = tripleCtrls[18] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy20 += pCtrl;
        ctrlxz20 += pCtrl;
        ctrlyz00 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 201
        pCase = tripleCases[19] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey0 += pCase;
        casez1 += pCase;
        casexy20 += pCase;
        casexz21 += pCase;
        caseyz01 += pCase;

        pCtrl = tripleCtrls[19] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy20 += pCtrl;
        ctrlxz21 += pCtrl;
        ctrlyz01 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 202
        pCase = tripleCases[20] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey0 += pCase;
        casez2 += pCase;
        casexy20 += pCase;
        casexz22 += pCase;
        caseyz02 += pCase;

        pCtrl = tripleCtrls[20] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly0 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy20 += pCtrl;
        ctrlxz22 += pCtrl;
        ctrlyz02 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 210
        pCase = tripleCases[21] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey1 += pCase;
        casez0 += pCase;
        casexy21 += pCase;
        casexz20 += pCase;
        caseyz10 += pCase;

        pCtrl = tripleCtrls[21] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy21 += pCtrl;
        ctrlxz20 += pCtrl;
        ctrlyz10 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 211
        pCase = tripleCases[22] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey1 += pCase;
        casez1 += pCase;
        casexy21 += pCase;
        casexz21 += pCase;
        caseyz11 += pCase;

        pCtrl = tripleCtrls[22] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy21 += pCtrl;
        ctrlxz21 += pCtrl;
        ctrlyz11 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 212
        pCase = tripleCases[23] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey1 += pCase;
        casez2 += pCase;
        casexy21 += pCase;
        casexz22 += pCase;
        caseyz12 += pCase;

        pCtrl = tripleCtrls[23] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly1 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy21 += pCtrl;
        ctrlxz22 += pCtrl;
        ctrlyz12 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 220
        pCase = tripleCases[24] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey2 += pCase;
        casez0 += pCase;
        casexy22 += pCase;
        casexz20 += pCase;
        caseyz20 += pCase;

        pCtrl = tripleCtrls[24] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz0 += pCtrl;
        ctrlxy22 += pCtrl;
        ctrlxz20 += pCtrl;
        ctrlyz20 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 221
        pCase = tripleCases[25] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey2 += pCase;
        casez1 += pCase;
        casexy22 += pCase;
        casexz21 += pCase;
        caseyz21 += pCase;

        pCtrl = tripleCtrls[25] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz1 += pCtrl;
        ctrlxy22 += pCtrl;
        ctrlxz21 += pCtrl;
        ctrlyz21 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }

        // 222
        pCase = tripleCases[26] * _invInds;
        if (pCase != 0.0) {
            entAll -= pCase * log2(pCase);
        }
        casex2 += pCase;
        casey2 += pCase;
        casez2 += pCase;
        casexy22 += pCase;
        casexz22 += pCase;
        caseyz22 += pCase;

        pCtrl = tripleCtrls[26] * _invInds;
        if (pCtrl != 0.0) {
            entAll -= pCtrl * log2(pCtrl);
        }
        ctrlx2 += pCtrl;
        ctrly2 += pCtrl;
        ctrlz2 += pCtrl;
        ctrlxy22 += pCtrl;
        ctrlxz22 += pCtrl;
        ctrlyz22 += pCtrl;

        pCase += pCtrl;
        if (pCase != 0.0) {
            entX -= pCase * log2(pCase);
        }


        float miX = _entY;
        if (casex0 != 0.0) {
            miX += casex0 * log2(casex0);
        }
        if (casex1 != 0.0) {
            miX += casex1 * log2(casex1);
        }
        if (casex2 != 0.0) {
            miX += casex2 * log2(casex2);
        }
        if (ctrlx0 != 0.0) {
            miX += ctrlx0 * log2(ctrlx0);
        }
        if (ctrlx1 != 0.0) {
            miX += ctrlx1 * log2(ctrlx1);
        }
        if (ctrlx2 != 0.0) {
            miX += ctrlx2 * log2(ctrlx2);
        }


        casex0 += ctrlx0;
        if (casex0 != 0.0) {
            miX -= casex0 * log2(casex0);
        }
        casex1 += ctrlx1;
        if (casex1 != 0.0) {
            miX -= casex1 * log2(casex1);
        }
        casex2 += ctrlx2;
        if (casex2 != 0.0) {
            miX -= casex2 * log2(casex2);
        }


        float miY = _entY;
        if (casey0 != 0.0) {
            miY += casey0 * log2(casey0);
        }
        if (casey1 != 0.0) {
            miY += casey1 * log2(casey1);
        }
        if (casey2 != 0.0) {
            miY += casey2 * log2(casey2);
        }
        if (ctrly0 != 0.0) {
            miY += ctrly0 * log2(ctrly0);
        }
        if (ctrly1 != 0.0) {
            miY += ctrly1 * log2(ctrly1);
        }
        if (ctrly2 != 0.0) {
            miY += ctrly2 * log2(ctrly2);
        }


        casey0 += ctrly0;
        if (casey0 != 0.0) {
            miY -= casey0 * log2(casey0);
        }
        casey1 += ctrly1;
        if (casey1 != 0.0) {
            miY -= casey1 * log2(casey1);
        }
        casey2 += ctrly2;
        if (casey2 != 0.0) {
            miY -= casey2 * log2(casey2);
        }


        float miZ = _entY;
        if (casez0 != 0.0) {
            miZ += casez0 * log2(casez0);
        }
        if (casez1 != 0.0) {
            miZ += casez1 * log2(casez1);
        }
        if (casez2 != 0.0) {
            miZ += casez2 * log2(casez2);
        }
        if (ctrlz0 != 0.0) {
            miZ += ctrlz0 * log2(ctrlz0);
        }
        if (ctrlz1 != 0.0) {
            miZ += ctrlz1 * log2(ctrlz1);
        }
        if (ctrlz2 != 0.0) {
            miZ += ctrlz2 * log2(ctrlz2);
        }


        casez0 += ctrlz0;
        if (casez0 != 0.0) {
            miZ -= casez0 * log2(casez0);
        }
        casez1 += ctrlz1;
        if (casez1 != 0.0) {
            miZ -= casez1 * log2(casez1);
        }
        casez2 += ctrlz2;
        if (casez2 != 0.0) {
            miZ -= casez2 * log2(casez2);
        }


        float igXY = _entY;
        if (casexy00 != 0.0) {
            igXY += casexy00 * log2(casexy00);
        }
        if (casexy01 != 0.0) {
            igXY += casexy01 * log2(casexy01);
        }
        if (casexy02 != 0.0) {
            igXY += casexy02 * log2(casexy02);
        }
        if (casexy10 != 0.0) {
            igXY += casexy10 * log2(casexy10);
        }
        if (casexy11 != 0.0) {
            igXY += casexy11 * log2(casexy11);
        }
        if (casexy12 != 0.0) {
            igXY += casexy12 * log2(casexy12);
        }
        if (casexy20 != 0.0) {
            igXY += casexy20 * log2(casexy20);
        }
        if (casexy21 != 0.0) {
            igXY += casexy21 * log2(casexy21);
        }
        if (casexy22 != 0.0) {
            igXY += casexy22 * log2(casexy22);
        }
        if (ctrlxy00 != 0.0) {
            igXY += ctrlxy00 * log2(ctrlxy00);
        }
        if (ctrlxy01 != 0.0) {
            igXY += ctrlxy01 * log2(ctrlxy01);
        }
        if (ctrlxy02 != 0.0) {
            igXY += ctrlxy02 * log2(ctrlxy02);
        }
        if (ctrlxy10 != 0.0) {
            igXY += ctrlxy10 * log2(ctrlxy10);
        }
        if (ctrlxy11 != 0.0) {
            igXY += ctrlxy11 * log2(ctrlxy11);
        }
        if (ctrlxy12 != 0.0) {
            igXY += ctrlxy12 * log2(ctrlxy12);
        }
        if (ctrlxy20 != 0.0) {
            igXY += ctrlxy20 * log2(ctrlxy20);
        }
        if (ctrlxy21 != 0.0) {
            igXY += ctrlxy21 * log2(ctrlxy21);
        }
        if (ctrlxy22 != 0.0) {
            igXY += ctrlxy22 * log2(ctrlxy22);
        }


        casexy00 += ctrlxy00;
        if (casexy00 != 0.0) {
            igXY -= casexy00 * log2(casexy00);
        }
        casexy01 += ctrlxy01;
        if (casexy01 != 0.0) {
            igXY -= casexy01 * log2(casexy01);
        }
        casexy02 += ctrlxy02;
        if (casexy02 != 0.0) {
            igXY -= casexy02 * log2(casexy02);
        }
        casexy10 += ctrlxy10;
        if (casexy10 != 0.0) {
            igXY -= casexy10 * log2(casexy10);
        }
        casexy11 += ctrlxy11;
        if (casexy11 != 0.0) {
            igXY -= casexy11 * log2(casexy11);
        }
        casexy12 += ctrlxy12;
        if (casexy12 != 0.0) {
            igXY -= casexy12 * log2(casexy12);
        }
        casexy20 += ctrlxy20;
        if (casexy20 != 0.0) {
            igXY -= casexy20 * log2(casexy20);
        }
        casexy21 += ctrlxy21;
        if (casexy21 != 0.0) {
            igXY -= casexy21 * log2(casexy21);
        }
        casexy22 += ctrlxy22;
        if (casexy22 != 0.0) {
            igXY -= casexy22 * log2(casexy22);
        }


        igXY -= miX + miY;
        if (igXY < 0.0) {
            igXY = 0.0;
        }


        float igXZ = _entY;
        if (casexz00 != 0.0) {
            igXZ += casexz00 * log2(casexz00);
        }
        if (casexz01 != 0.0) {
            igXZ += casexz01 * log2(casexz01);
        }
        if (casexz02 != 0.0) {
            igXZ += casexz02 * log2(casexz02);
        }
        if (casexz10 != 0.0) {
            igXZ += casexz10 * log2(casexz10);
        }
        if (casexz11 != 0.0) {
            igXZ += casexz11 * log2(casexz11);
        }
        if (casexz12 != 0.0) {
            igXZ += casexz12 * log2(casexz12);
        }
        if (casexz20 != 0.0) {
            igXZ += casexz20 * log2(casexz20);
        }
        if (casexz21 != 0.0) {
            igXZ += casexz21 * log2(casexz21);
        }
        if (casexz22 != 0.0) {
            igXZ += casexz22 * log2(casexz22);
        }
        if (ctrlxz00 != 0.0) {
            igXZ += ctrlxz00 * log2(ctrlxz00);
        }
        if (ctrlxz01 != 0.0) {
            igXZ += ctrlxz01 * log2(ctrlxz01);
        }
        if (ctrlxz02 != 0.0) {
            igXZ += ctrlxz02 * log2(ctrlxz02);
        }
        if (ctrlxz10 != 0.0) {
            igXZ += ctrlxz10 * log2(ctrlxz10);
        }
        if (ctrlxz11 != 0.0) {
            igXZ += ctrlxz11 * log2(ctrlxz11);
        }
        if (ctrlxz12 != 0.0) {
            igXZ += ctrlxz12 * log2(ctrlxz12);
        }
        if (ctrlxz20 != 0.0) {
            igXZ += ctrlxz20 * log2(ctrlxz20);
        }
        if (ctrlxz21 != 0.0) {
            igXZ += ctrlxz21 * log2(ctrlxz21);
        }
        if (ctrlxz22 != 0.0) {
            igXZ += ctrlxz22 * log2(ctrlxy22);
        }

        casexz00 += ctrlxz00;
        if (casexz00 != 0.0) {
            igXZ -= casexz00 * log2(casexz00);
        }
        casexz01 += ctrlxz01;
        if (casexz01 != 0.0) {
            igXZ -= casexz01 * log2(casexz01);
        }
        casexz02 += ctrlxz02;
        if (casexz02 != 0.0) {
            igXZ -= casexz02 * log2(casexz02);
        }
        casexz10 += ctrlxz10;
        if (casexz10 != 0.0) {
            igXZ -= casexz10 * log2(casexz10);
        }
        casexz11 += ctrlxz11;
        if (casexz11 != 0.0) {
            igXZ -= casexz11 * log2(casexz11);
        }
        casexz12 += ctrlxz12;
        if (casexz12 != 0.0) {
            igXZ -= casexz12 * log2(casexz12);
        }
        casexz20 += ctrlxz20;
        if (casexz20 != 0.0) {
            igXZ -= casexz20 * log2(casexz20);
        }
        casexz21 += ctrlxz21;
        if (casexz21 != 0.0) {
            igXZ -= casexz21 * log2(casexz21);
        }
        casexz22 += ctrlxz22;
        if (casexz22 != 0.0) {
            igXZ -= casexz22 * log2(casexz22);
        }

        igXZ -= miX + miZ;
        if (igXZ < 0.0) {
            igXZ = 0.0;
        }

        float igYZ = _entY;
        if (caseyz00 != 0.0) {
            igYZ += caseyz00 * log2(caseyz00);
        }
        if (caseyz01 != 0.0) {
            igYZ += caseyz01 * log2(caseyz01);
        }
        if (caseyz02 != 0.0) {
            igYZ += caseyz02 * log2(caseyz02);
        }
        if (caseyz10 != 0.0) {
            igYZ += caseyz10 * log2(caseyz10);
        }
        if (caseyz11 != 0.0) {
            igYZ += caseyz11 * log2(caseyz11);
        }
        if (caseyz12 != 0.0) {
            igYZ += caseyz12 * log2(caseyz12);
        }
        if (caseyz20 != 0.0) {
            igYZ += caseyz20 * log2(caseyz20);
        }
        if (caseyz21 != 0.0) {
            igYZ += caseyz21 * log2(caseyz21);
        }
        if (caseyz22 != 0.0) {
            igYZ += caseyz22 * log2(caseyz22);
        }
        if (ctrlyz00 != 0.0) {
            igYZ += ctrlyz00 * log2(ctrlyz00);
        }
        if (ctrlyz01 != 0.0) {
            igYZ += ctrlyz01 * log2(ctrlyz01);
        }
        if (ctrlyz02 != 0.0) {
            igYZ += ctrlyz02 * log2(ctrlyz02);
        }
        if (ctrlyz10 != 0.0) {
            igYZ += ctrlyz10 * log2(ctrlyz10);
        }
        if (ctrlyz11 != 0.0) {
            igYZ += ctrlyz11 * log2(ctrlyz11);
        }
        if (ctrlyz12 != 0.0) {
            igYZ += ctrlyz12 * log2(ctrlyz12);
        }
        if (ctrlyz20 != 0.0) {
            igYZ += ctrlyz20 * log2(ctrlyz20);
        }
        if (ctrlyz21 != 0.0) {
            igYZ += ctrlyz21 * log2(ctrlyz21);
        }
        if (ctrlyz22 != 0.0) {
            igYZ += ctrlyz22 * log2(ctrlyz22);
        }

        caseyz00 += ctrlyz00;
        if (caseyz00 != 0.0) {
            igYZ -= caseyz00 * log2(caseyz00);
        }
        caseyz01 += ctrlyz01;
        if (caseyz01 != 0.0) {
            igYZ -= caseyz01 * log2(caseyz01);
        }
        caseyz02 += ctrlyz02;
        if (caseyz02 != 0.0) {
            igYZ -= caseyz02 * log2(caseyz02);
        }
        caseyz10 += ctrlyz10;
        if (caseyz10 != 0.0) {
            igYZ -= caseyz10 * log2(caseyz10);
        }
        caseyz11 += ctrlyz11;
        if (caseyz11 != 0.0) {
            igYZ -= caseyz11 * log2(caseyz11);
        }
        caseyz12 += ctrlyz12;
        if (caseyz12 != 0.0) {
            igYZ -= caseyz12 * log2(caseyz12);
        }
        caseyz20 += ctrlyz20;
        if (caseyz20 != 0.0) {
            igYZ -= caseyz20 * log2(caseyz20);
        }
        caseyz21 += ctrlyz21;
        if (caseyz21 != 0.0) {
            igYZ -= caseyz21 * log2(caseyz21);
        }
        caseyz22 += ctrlyz22;
        if (caseyz22 != 0.0) {
            igYZ -= caseyz22 * log2(caseyz22);
        }

        igYZ -= miZ + miY;
        if (igYZ < 0.0) {
            igYZ = 0.0;
        }

        miXYZ = _entY + entX - entAll;

        //printf("Thread %d in block %d: miXYZ for triple (%u, %u, %u) is %f (%f+%f-%f)\n", threadIdx.x,
        //	blockIdx.x, myId1, myId2, iterId3, miXYZ, _entY, entX, entAll);

        entX = miXYZ - igXY - igXZ - igYZ - miX - miY - miZ;

#ifdef DEBUG
        printf("Thread %d in block %d: IG for triple (%u, %u, %u) is %f (%f-%f-%f-%f-%f-%f-%f)\n", threadIdx.x,
                blockIdx.x, myId1, myId2, iterId3, entX, miXYZ, igXY, igXZ, miX, miY, miZ);
#endif


        // Now include the value in the output list if it is high enough
        // There are empty values in the array
        if (numEntriesWithMI < numOutputs) {
            myOutIds[numEntriesWithMI] = iterId3;
            myOutValues[numEntriesWithMI] = entX;

            // If this is the minimum value of the array
            if (entX < minMI) {
                minMI = entX;
                minMIPos = numEntriesWithMI;
            }

            numEntriesWithMI++;
        } else if (entX > minMI) { // The value must be inserted
            myOutIds[minMIPos] = iterId3;
            myOutValues[minMIPos] = entX;

            // Find the new minimum
            minMIPos = 0;
            minMI = myOutIds[0];
            for (int i = 1; i < numOutputs; i++) {
                if (myOutValues[i] < minMI) {
                    minMI = myOutValues[i];
                    minMIPos = i;
                }
            }
        }
    }

    // The thread has a list of numOutputs with the highest values
    // Complete the list just in case there are no so many values in total
    for (int i = numEntriesWithMI; i < numOutputs; i++) {
        myOutIds[i] = 0;
        myOutValues[i] = 0.0;
        minMI = 0.0;
        minMIPos = i;
    }

#ifdef DEBUG
    printf("Before reducing thread %d of block %d: %u (%f), %u (%f), %u (%f), %u (%f)\n",
                threadIdx.x, blockIdx.x,
                myOutIds[0], myOutValues[0], myOutIds[1], myOutValues[1],
                myOutIds[2], myOutValues[2], myOutIds[3], myOutValues[3]);
#endif

    float *remoteOutValues;
    uint32_t * remoteOutIds;

    // Perform the reduction of the lists of the block of threads
    // Each reduction obtains the numOutputs highest elements of two threads
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        __syncthreads();
        if (threadIdx.x < stride) {
            // Each thread has its own minimum so we only need to compare the numOutputs values of the other thread
            remoteOutValues = &shMIValues[(threadIdx.x + stride) * numOutputs];
            remoteOutIds = &shMIId[(threadIdx.x + stride) * numOutputs];

            for (int i = 0; i < numOutputs; i++) {
                if (remoteOutValues[i] > minMI) { // The value must be inserted
                    myOutIds[minMIPos] = remoteOutIds[i];
                    myOutValues[minMIPos] = remoteOutValues[i];

                    // Find the new minimum
                    minMIPos = 0;
                    minMI = myOutIds[0];
                    for (int j = 1; j < numOutputs; j++) {
                        if (myOutValues[j] < minMI) {
                            minMI = myOutValues[j];
                            minMIPos = j;
                        }
                    }
                }
            }
#ifdef DEBUG
            printf("With stride %d thread %d of block %d: %u (%f), %u (%f), %u (%f), %u (%f)\n", stride,
                threadIdx.x, blockIdx.x,
                myOutIds[0], myOutValues[0], myOutIds[1], myOutValues[1],
                myOutIds[2], myOutValues[2], myOutIds[3], myOutValues[3]);
#endif
        }
    }

    // Save the output list for the block
    if (!threadIdx.x) {
        uint3 *blockOutIds = &devMiIds[blockIdx.x * numOutputs];
        float *blockOutValues = &devMIValues[blockIdx.x * numOutputs];

        for (int i = 0; i < numOutputs; i++) {
            blockOutIds[i].x = myId1;
            blockOutIds[i].y = myId2;
            blockOutIds[i].z = myOutIds[i];
            blockOutValues[i] = myOutValues[i];
        }
    }
}

MutualInformation::MutualInformation(bool isMI, uint32_t numSNPs, uint16_t numCases, uint16_t numCtrls,
                                     std::vector<std::vector<uint32_t> *> cases,
                                     std::vector<std::vector<uint32_t> *> ctrls) :
        _isMI(isMI),
        _numSNPs(numSNPs),
        _numEntriesCase(numCases / 32 + ((numCases % 32) > 0)),
        _numEntriesCtrl(numCtrls / 32 + ((numCtrls % 32) > 0)) {
    // Allocate the arrays
    if (hipSuccess != hipMalloc(&_dev0Cases, _numEntriesCase * _numSNPs * sizeof(uint32_t)))
        throw CUDAError();
    if (hipSuccess != hipMalloc(&_dev1Cases, _numEntriesCase * _numSNPs * sizeof(uint32_t)))
        throw CUDAError();
    if (hipSuccess != hipMalloc(&_dev2Cases, _numEntriesCase * _numSNPs * sizeof(uint32_t)))
        throw CUDAError();
    if (hipSuccess != hipMalloc(&_dev0Ctrls, _numEntriesCtrl * _numSNPs * sizeof(uint32_t)))
        throw CUDAError();
    if (hipSuccess != hipMalloc(&_dev1Ctrls, _numEntriesCtrl * _numSNPs * sizeof(uint32_t)))
        throw CUDAError();
    if (hipSuccess != hipMalloc(&_dev2Ctrls, _numEntriesCtrl * _numSNPs * sizeof(uint32_t)))
        throw CUDAError();

    // All the entries are cyclicly ordered by SNPs
    if (hipSuccess !=
        hipMemcpy(_dev0Cases, &cases[0][0][0], _numSNPs * _numEntriesCase * sizeof(uint32_t), hipMemcpyHostToDevice))
        throw CUDAError();
    if (hipSuccess !=
        hipMemcpy(_dev1Cases, &cases[0][1][0], _numSNPs * _numEntriesCase * sizeof(uint32_t), hipMemcpyHostToDevice))
        throw CUDAError();
    if (hipSuccess !=
        hipMemcpy(_dev2Cases, &cases[0][2][0], _numSNPs * _numEntriesCase * sizeof(uint32_t), hipMemcpyHostToDevice))
        throw CUDAError();
    if (hipSuccess !=
        hipMemcpy(_dev0Ctrls, &ctrls[0][0][0], _numSNPs * _numEntriesCtrl * sizeof(uint32_t), hipMemcpyHostToDevice))
        throw CUDAError();
    if (hipSuccess !=
        hipMemcpy(_dev1Ctrls, &ctrls[0][1][0], _numSNPs * _numEntriesCtrl * sizeof(uint32_t), hipMemcpyHostToDevice))
        throw CUDAError();
    if (hipSuccess !=
        hipMemcpy(_dev2Ctrls, &ctrls[0][2][0], _numSNPs * _numEntriesCtrl * sizeof(uint32_t), hipMemcpyHostToDevice))
        throw CUDAError();

    // Increase the size of the L1 compared to shared memory
    if (hipSuccess != hipFuncSetCacheConfig(reinterpret_cast<const void*>(_kernelDoubleTable), hipFuncCachePreferL1))
        throw CUDAError();

    // Increase the size of the shared memory compared to L1
    if (hipSuccess != hipFuncSetCacheConfig(reinterpret_cast<const void*>(_kernelTripleMI), hipFuncCachePreferShared))
        throw CUDAError();

    // Increase the size of the shared memory compared to L1
    if (hipSuccess != hipFuncSetCacheConfig(reinterpret_cast<const void*>(_kernelTripleIG), hipFuncCachePreferShared))
        throw CUDAError();

    float invInds = 1.0 / (numCases + numCtrls);
    float p = numCases * invInds;
    float entY = (-1.0) * p * log2(p);

    p = numCtrls * invInds;
    entY -= p * log2(p);

    if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(_invInds), &invInds, sizeof(float), 0, hipMemcpyHostToDevice))
        throw CUDAError();

    if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(_entY), &entY, sizeof(float), 0, hipMemcpyHostToDevice))
        throw CUDAError();

    float maxFL = FLT_MAX;
    if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(_MAX_FLOAT), &maxFL, sizeof(float), 0, hipMemcpyHostToDevice))
        throw CUDAError();
}

MutualInformation::~MutualInformation() {
    if (hipSuccess != hipFree(_dev0Cases))
        throw CUDAError();
    if (hipSuccess != hipFree(_dev1Cases))
        throw CUDAError();
    if (hipSuccess != hipFree(_dev2Cases))
        throw CUDAError();
    if (hipSuccess != hipFree(_dev0Ctrls))
        throw CUDAError();
    if (hipSuccess != hipFree(_dev1Ctrls))
        throw CUDAError();
    if (hipSuccess != hipFree(_dev2Ctrls))
        throw CUDAError();
}

long MutualInformation::compute(const std::vector<uint2> &pairs, uint16_t num_outputs, Position *output) {
    constexpr size_t block_size = 5000;

    if (hipSuccess != hipMalloc(&_devIds, block_size * sizeof(uint2)))
        throw CUDAError();

    // Auxiliary array for the contingency tables between the two kernels
    GPUDoubleContTable *_tables = new GPUDoubleContTable[block_size];
    for (int i = 0; i < block_size; i++) {
        _tables[i].initialize(_numEntriesCase, _numEntriesCtrl);
    }

    GPUDoubleContTable *_devDoubleTables;
    if (hipSuccess != hipMalloc(&_devDoubleTables, block_size * sizeof(GPUDoubleContTable)))
        throw CUDAError();
    if (hipSuccess !=
        hipMemcpy(_devDoubleTables, _tables, block_size * sizeof(GPUDoubleContTable),
                   hipMemcpyHostToDevice))
        throw CUDAError();

    // Auxiliary array to store the MI values of each block
    float *_devMIValues;
    if (hipSuccess != hipMalloc(&_devMIValues, block_size * num_outputs * sizeof(float)))
        throw CUDAError();
    float *_hostMIValues = new float[block_size * num_outputs];

    // Auxiliary arrays to store the ids that are in the list of MIs
    uint3 *_devMiIds;
    if (hipSuccess != hipMalloc(&_devMiIds, block_size * num_outputs * sizeof(uint3)))
        throw CUDAError();
    uint3 *_hostMiIds = new uint3[block_size * num_outputs];

    // The minimum value in the array
    float minMI = FLT_MAX;
    // The position of the minimum value
    uint16_t minMIPos = 0;
    // Number of entries of the array full
    uint16_t numEntriesWithMI = 0;

    for (unsigned long i = 0; i < pairs.size(); i += block_size) {
        const unsigned long num_pairs = pairs.size() - i < block_size ? pairs.size() - i : block_size;

        if (hipSuccess != hipMemcpy(_devIds, &pairs.at(0) + i, num_pairs * sizeof(uint2), hipMemcpyHostToDevice))
            throw CUDAError();

        uint32_t
        nblocks = (num_pairs + NUM_TH_PER_BLOCK - 1) / NUM_TH_PER_BLOCK;
        dim3 gridDouble(nblocks, 1);
        dim3 blocksDouble(NUM_TH_PER_BLOCK, 1);

        // Starts computing the double contingency tables
        _kernelDoubleTable << < gridDouble, blocksDouble, 0 >> > (num_pairs, _numSNPs,
                _numEntriesCase, _numEntriesCtrl, _devIds,
                _dev0Cases, _dev1Cases, _dev2Cases, _dev0Ctrls, _dev1Ctrls, _dev2Ctrls,
                _devDoubleTables);

        // Now you need to calculate the MI for each triple
        // Each block performs all the triples for one pair
        // The necessary shared memory is to store the double contingency table of the matrix
        dim3 gridMI(num_pairs, 1);
        dim3 blockMI(NUM_TH_PER_BLOCK, 1);
        uint32_t
        sharedSize = 9 * (_numEntriesCase + _numEntriesCtrl) * sizeof(uint32_t);
        sharedSize += num_outputs * NUM_TH_PER_BLOCK * (sizeof(float) + sizeof(uint32_t));

        if (_isMI) {
            _kernelTripleMI << < gridMI, blockMI, sharedSize >> >
                                                  (num_pairs, _numSNPs, _numEntriesCase, _numEntriesCtrl,
                                                          _devIds, _dev0Cases, _dev1Cases, _dev2Cases, _dev0Ctrls, _dev1Ctrls, _dev2Ctrls,
                                                          _devDoubleTables, num_outputs, _devMIValues, _devMiIds);
        } else {
            _kernelTripleIG << < gridMI, blockMI, sharedSize >> >
                                                  (num_pairs, _numSNPs, _numEntriesCase, _numEntriesCtrl,
                                                          _devIds, _dev0Cases, _dev1Cases, _dev2Cases, _dev0Ctrls, _dev1Ctrls, _dev2Ctrls,
                                                          _devDoubleTables, num_outputs, _devMIValues, _devMiIds);
        }

        if (hipSuccess != hipMemcpy(_hostMIValues, _devMIValues, num_pairs * num_outputs * sizeof(float),
                                      hipMemcpyDeviceToHost))
            throw CUDAError();
        if (hipSuccess != hipMemcpy(_hostMiIds, _devMiIds, num_pairs * num_outputs * sizeof(uint3),
                                      hipMemcpyDeviceToHost))
            throw CUDAError();

        _findNHighestMI(_hostMiIds, _hostMIValues, num_pairs * num_outputs, minMI, minMIPos, numEntriesWithMI,
                        num_outputs, output);
    }

    if (hipSuccess != hipFree(_devMIValues))
        throw CUDAError();
    if (hipSuccess != hipFree(_devMiIds))
        throw CUDAError();
    if (hipSuccess != hipFree(_devIds))
        throw CUDAError();
    if (hipSuccess != hipFree(_devDoubleTables))
        throw CUDAError();

    for (int i = 0; i < block_size; i++) {
        _tables[i].finalize();
    }
    delete[] _tables;
    delete[] _hostMIValues;
    delete[] _hostMiIds;

    long myTotalAnal = 0;
    for (auto p : pairs) {
        myTotalAnal += _numSNPs - p.y - 1;
    }
    return myTotalAnal;
}

void MutualInformation::_findNHighestMI(uint3 *_hostMiIds, float *_hostMIValues, uint64_t totalValues, float &minMI,
                                        uint16_t &minMIPos, uint16_t &numEntriesWithMI,
                                        size_t num_outputs, Position *output) {
    int iter = 0;
    Position *auxMI;
    float auxValue;

    if (numEntriesWithMI == 0) { // The first values are directly stored
        for (iter = 0; iter < num_outputs; iter++) {
            auxValue = _hostMIValues[iter];

            auxMI = &output[iter];
            auxMI->rank = auxValue;
            auxMI->p1 = _hostMiIds[iter].x;
            auxMI->p2 = _hostMiIds[iter].y;
            auxMI->p3 = _hostMiIds[iter].z;

            if (auxValue < minMI) {
                minMI = auxValue;
                minMIPos = iter;
            }
        }
        numEntriesWithMI += num_outputs;
    }

    for (; iter < totalValues; iter++) {
        auxValue = _hostMIValues[iter];

        if (auxValue > minMI) { // The value must be inserted
            auxMI = &output[minMIPos];
            auxMI->p1 = _hostMiIds[iter].x;
            auxMI->p2 = _hostMiIds[iter].y;
            auxMI->p3 = _hostMiIds[iter].z;
            auxMI->rank = auxValue;

            // Find the new minimum
            auxMI = std::min_element(output, output + num_outputs);
            minMI = auxMI->rank;
            uint16_t i = 0;
            while (1) {
                if (output[i].rank == minMI) {
                    break;
                }
                i++;
            }
            minMIPos = i;
        }
    }
}
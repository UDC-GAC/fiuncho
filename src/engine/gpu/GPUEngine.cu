#include "hip/hip_runtime.h"
/*
 * MPI3SNP ~ https://github.com/chponte/mpi3snp
 *
 * Copyright 2018 Christian Ponte
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated
 * documentation files (the "Software"), to deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all copies or substantial portions of the
 * Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE
 * WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
 * @file gpu/GPUEngine.cpp
 * @author Christian Ponte
 * @date 1 March 2018
 *
 * @brief GPUEngine class implementation.
 */

#include "GPUEngine.h"
#include "Distributor.h"
#include "MutualInformation.h"
#include <cstring>

GPUEngine::GPUEngine(unsigned int proc_num, unsigned int proc_id,
                     std::vector<std::pair<unsigned int, unsigned int>> gpu_map, bool use_mi, Statistics &statistics) :
        proc_num(proc_num),
        proc_id(proc_id),
        use_mi(use_mi),
        statistics(statistics) {
    statistics.Begin_timer("CUDA initialization time");
    hipFree(nullptr);
    statistics.End_timer("CUDA initialization time");

    int avail_gpus = 0;
    if (hipSuccess != hipGetDeviceCount(&avail_gpus))
        throw CUDAError();
    if (avail_gpus == 0) {
        throw CUDAError("Could not find any CUDA-enabled GPU");
    }

    auto pos = std::find_if(gpu_map.begin(), gpu_map.end(),
                            [&proc_id](std::pair<unsigned int, unsigned int> item) { return item.first == proc_id; });
    gpu_id = pos == gpu_map.end() ? proc_id % avail_gpus : pos->second;

    hipDeviceProp_t gpu_prop;
    if (hipSuccess != hipGetDeviceProperties(&gpu_prop, gpu_id))
        throw CUDAError();
    if (gpu_prop.major < 2 || !gpu_prop.canMapHostMemory) {
        throw CUDAError("GPU " + std::to_string(gpu_id) + " does not meet compute capabilities\n" +
                        "Name: " + gpu_prop.name + "\n" + "Compute capability: " +
                        std::to_string(gpu_prop.major) + "." + std::to_string(gpu_prop.minor));
    }
    if (hipSuccess != hipSetDevice(gpu_id))
        throw CUDAError();
}

void GPUEngine::run(std::string tped, std::string tfam, std::vector<Position> &output, size_t num_outputs) {
    statistics.Begin_timer("SNPs read time");
    Dataset *dataset;
    try {
        dataset = new Dataset(tped, tfam, Dataset::Transposed);
    } catch (const Dataset::Read_error &error) {
        throw Engine::Error(error.what());
    }
    statistics.End_timer("SNPs read time");

    statistics.Addi("SNP count", dataset->get_SNP_count());
    statistics.Addi("Number of cases", dataset->get_case_count());
    statistics.Addi("Number of controls", dataset->get_ctrl_count());

    Distributor<uint32_t, uint2> distributor(dataset->get_SNP_count(), proc_num);

    Algorithm<uint2> *search = new MutualInformation(use_mi, dataset->get_SNP_count(), dataset->get_case_count(), dataset->get_ctrl_count(),
                         dataset->get_cases(), dataset->get_ctrls());

    std::vector<uint2> pairs;
    distributor.get_pairs([](uint32_t x, uint32_t y) {
        uint2 p {x, y};
        return p;
    }, proc_id, pairs);

    std::string timer_label;
    timer_label += "GPU " + std::to_string(gpu_id) + " runtime";
    statistics.Begin_timer(timer_label);

    output.resize(num_outputs);
    long myTotalAnal = search->compute(pairs, num_outputs, &output.at(0));
    hipDeviceSynchronize();
    statistics.Addl("GPU " + std::to_string(gpu_id) + " computations", myTotalAnal);

    delete search;
    delete dataset;

    statistics.End_timer(timer_label);
}